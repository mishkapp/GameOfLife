
#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C"
__device__ int getNeighborhoods(int i, int j, int maxI, int maxJ, short *pixels){

        int neighborhoods = 0;
        int width = maxI+1;

        if(i != 0 && i != maxI && j != 0 && j != maxJ){
            if(pixels[i + (j-1) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i + (j+1) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i-1 + (j-1) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i-1 + (j) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i-1 + (j+1) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i+1 + (j-1) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i+1 + (j) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i+1 + (j+1) * width] == 1){
                neighborhoods +=1;
            }
        }


        if(j == 0 && i != 0 && i != maxI){
            if(pixels[i-1 + (j) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i+1 + (j) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i-1 + (j+1) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i + (j+1) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i+1 + (j+1) * width] == 1){
                neighborhoods +=1;
            }
            if(true){
                if(pixels[i-1 + (maxJ) * width] == 1){
                    neighborhoods +=1;
                }
                if(pixels[i + (maxJ) * width] == 1){
                    neighborhoods +=1;
                }
                if(pixels[i+1 + (maxJ) * width] == 1){
                    neighborhoods +=1;
                }
            }
        }
        if(j == maxJ && i != 0 && i != maxI){
            if(pixels[i-1 + (j) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i+1 + (j) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i-1 + (j-1) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i + (j-1) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i+1 + (j-1) * width] == 1){
                neighborhoods +=1;
            }
            if(true){
                if(pixels[i-1 + (0) * width] == 1){
                    neighborhoods +=1;
                }
                if(pixels[i + (0) * width] == 1){
                    neighborhoods +=1;
                }
                if(pixels[i+1 + (0) * width] == 1){
                    neighborhoods +=1;
                }
            }
        }
        if(i == 0 && j != 0 && j != maxJ){
            if(pixels[i + (j+1) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i + (j-1) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i+1 + (j-1) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i+1 + (j) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i+1 + (j+1) * width] == 1){
                neighborhoods +=1;
            }
            if(true){
                if(pixels[maxI + (j-1) * width] == 1){
                    neighborhoods +=1;
                }
                if(pixels[maxI + (j) * width] == 1){
                    neighborhoods +=1;
                }
                if(pixels[maxI + (j+1) * width] == 1){
                    neighborhoods +=1;
                }
            }
        }
        if(i == maxI && j != 0 && j != maxJ){
            if(pixels[i + (j+1) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i + (j-1) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i-1 + (j-1) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i-1 + (j) * width] == 1){
                neighborhoods +=1;
            }
            if(pixels[i-1 + (j+1) * width] == 1){
                neighborhoods +=1;
            }
            if(true){
                if(pixels[0 + (j-1) * width] == 1){
                    neighborhoods +=1;
                }
                if(pixels[0 + (j) * width] == 1){
                    neighborhoods +=1;
                }
                if(pixels[0 + (j+1) * width] == 1){
                    neighborhoods +=1;
                }
            }
        }
        //CORNERS
        if(i == 0 && j == 0){
            if(pixels[i+1 + (j) * width] == 1){
                neighborhoods += 1;
            }
            if(pixels[i+1 + (j+1) * width] == 1){
                neighborhoods += 1;
            }
            if(pixels[i + (j+1) * width] == 1){
                neighborhoods += 1;
            }
            if(true){
                if(pixels[i+1 + (maxJ) * width] == 1){
                    neighborhoods += 1;
                }
                if(pixels[i + (maxJ) * width] == 1){
                    neighborhoods += 1;
                }
                if(pixels[maxI + (j) * width] == 1){
                    neighborhoods += 1;
                }
                if(pixels[maxI + (j+1) * width] == 1){
                    neighborhoods += 1;
                }
                if(pixels[maxI + (maxJ) * width] == 1){
                    neighborhoods += 1;
                }
            }
        }

        if(i == maxI && j == 0){
            if(pixels[i-1 + (j) * width] == 1){
                neighborhoods += 1;
            }
            if(pixels[i-1 + (j+1) * width] == 1){
                neighborhoods += 1;
            }
            if(pixels[i + (j+1) * width] == 1){
                neighborhoods += 1;
            }
            if(true){
                if(pixels[i-1 + (maxJ) * width] == 1){
                    neighborhoods += 1;
                }
                if(pixels[i + (maxJ) * width] == 1){
                    neighborhoods += 1;
                }
                if(pixels[0 + (j) * width] == 1){
                    neighborhoods += 1;
                }
                if(pixels[0 + (j+1) * width] == 1){
                    neighborhoods += 1;
                }
                if(pixels[0 + (maxJ) * width] == 1){
                    neighborhoods += 1;
                }
            }
        }

        if(i == maxI && j == maxJ){
            if(pixels[i-1 + (j) * width] == 1){
                neighborhoods += 1;
            }
            if(pixels[i-1 + (j-1) * width] == 1){
                neighborhoods += 1;
            }
            if(pixels[i + (j-1) * width] == 1){
                neighborhoods += 1;
            }
            if(true){
                if(pixels[i-1 + (0) * width] == 1){
                    neighborhoods += 1;
                }
                if(pixels[i + (0) * width] == 1){
                    neighborhoods += 1;
                }
                if(pixels[0 + (j) * width] == 1){
                    neighborhoods += 1;
                }
                if(pixels[0 + (j-1) * width] == 1){
                    neighborhoods += 1;
                }
                if(pixels[0 + (0) * width] == 1){
                    neighborhoods += 1;
                }
            }
        }
        if(i == 0 && j == maxJ){
            if(pixels[i+1 + (j) * width] == 1){
                neighborhoods += 1;
            }
            if(pixels[i+1 + (j-1) * width] == 1){
                neighborhoods += 1;
            }
            if(pixels[i + (j-1) * width] == 1){
                neighborhoods += 1;
            }
            if(true){
                if(pixels[i+1 + (0) * width] == 1){
                    neighborhoods += 1;
                }
                if(pixels[i + (0) * width] == 1){
                    neighborhoods += 1;
                }
                if(pixels[maxI + (j) * width] == 1){
                    neighborhoods += 1;
                }
                if(pixels[maxI + (j-1) * width] == 1){
                    neighborhoods += 1;
                }
                if(pixels[maxI] == 1){
                    neighborhoods += 1;
                }
            }
        }
        //END CORNERS
        return neighborhoods;
    }
extern "C"
__global__ void handle(short *sourcePixels, short *newPixels, int width, int height, int echelon){
    int i = threadIdx.x;
    int j = i / width;

    int neighborhoods = getNeighborhoods(i,j,width-1,height-1, sourcePixels);
    if(sourcePixels[i] == 1){
        if(neighborhoods >= 2 && neighborhoods <= 3){
            newPixels[i] = 1;
        }else{
            newPixels[i] = 0;
        }
    }else{
        if(neighborhoods == 3){
            newPixels[i] = 1;
        }else{
            newPixels[i] = 0;
        }
    }
}
// extern "C"
// __host__ void start(short *sourcePixels, short *newPixels, int *width, int *height, int *echelon){

// 	int N = *height * *width;

// 	int *d_width;
// 	int *d_height;
// 	int *d_echelon;

//     short *d_sourcePixels;
//     short *d_newPixels;

// 	cudaMalloc((void **)&d_width, sizeof(int));
// 	cudaMalloc((void **)&d_height, sizeof(int));
// 	cudaMalloc((void **)&d_echelon, sizeof(int)); //??
//     cudaMalloc((void **)&d_sourcePixels, sizeof(short) * N);
//     cudaMalloc((void **)&d_newPixels, sizeof(short) * N);
    

// 	cudaMemcpy(d_width, &width, sizeof(int), cudaMemcpyHostToDevice);
// 	cudaMemcpy(d_height, &height, sizeof(int), cudaMemcpyHostToDevice);
// 	cudaMemcpy(d_echelon, &echelon, sizeof(int), cudaMemcpyHostToDevice);
//     cudaMemcpy(d_sourcePixels, &sourcePixels, sizeof(short) * N, cudaMemcpyHostToDevice);
//     cudaMemcpy(d_newPixels, &sourcePixels, sizeof(short) * N, cudaMemcpyHostToDevice);

// 	handle<<<1,N>>>(d_sourcePixels, d_newPixels, *d_width, *d_height, *d_echelon);

//     cudaMemcpy(newPixels, &d_newPixels, sizeof(short) * N, cudaMemcpyDeviceToHost);
//     // cudaFree();
// }

